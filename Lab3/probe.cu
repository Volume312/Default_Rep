#include <stdio.h>

int main(int argc, char const *argv[])
{
	//Alibek Cholponbaev Assignment 3
	//code from the book

	//get  device count
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	printf("number of GPU devices: %d\n\n", count);

	//get device props
	hipDeviceProp_t prop;
	for(int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

		//from the book (from your file)
		printf("\tName: %s\n", prop.name);
	    printf( "\tCompute capability: %d.%d\n", prop.major, prop.minor);
	    printf( "\tClock rate: %d\n", prop.clockRate );
	    printf( "\tDevice copy overlap: " );
	      if (prop.deviceOverlap)
	        printf( "Enabled\n" );
	      else
	        printf( "Disabled\n" );
	    printf( "\tKernel execition timeout: " );
	      if (prop.kernelExecTimeoutEnabled)
	        printf( "Enabled\n" );
	      else
	        printf( "Disabled\n" );
	    printf( "--- Memory Information for device %d ---\n", i );
	    printf("\tTotal global mem: %ld\n", prop.totalGlobalMem );
	    printf("\tTotal constant Mem: %ld\n", prop.totalConstMem );
	    printf("\tMax mem pitch: %ld\n", prop.memPitch );
	    printf( "\tTexture Alignment: %ld\n", prop.textureAlignment );
	    printf("\n");
	    printf( "\tMultiprocessor count: %d\n", prop.multiProcessorCount );
	    printf( "\tShared mem per processor: %ld\n", prop.sharedMemPerBlock );
	    printf( "\tRegisters per processor: %d\n", prop.regsPerBlock );
	    printf( "\tThreads in warp: %d\n", prop.warpSize );
	    printf( "\tMax threads per block: %d\n", prop.maxThreadsPerBlock );
	    printf( "\tMax block dimensions: (%d, %d, %d)\n",
	                  prop.maxThreadsDim[0],
	                  prop.maxThreadsDim[1],
	                  prop.maxThreadsDim[2]);
	    printf( "\tMax grid dimensions: (%d, %d, %d)\n",
	                  prop.maxGridSize[0],
	                  prop.maxGridSize[1],
	                  prop.maxGridSize[2]);
	    printf("\n");
	}
	return 0;
}